#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <iostream>
#include <fstream>
#include <time.h>
#include <hip/hip_runtime.h>
#include <hip/hip_cooperative_groups.h>

#define smooth_kernel_size 5
#define sigma 1.0
#define K  1.0

#define WIDTH1 480
#define WIDTH2 360

using namespace std;


__global__ void convolution_2D(int N[WIDTH1*WIDTH2], double M[smooth_kernel_size], int P[WIDTH1*WIDTH2], double T[WIDTH1*WIDTH2]) {
    //cooperative_groups::grid_group grid = cooperative_groups::this_grid();

    // find center position of kernel (half of kernel size)
    int kCenterX = smooth_kernel_size / 2;
    int kCenterY = smooth_kernel_size / 2;

    int i = blockIdx.x;
    int j = threadIdx.x;
    //printf("%d %d\n", gridDim.x, blockDim.x);

    T[i*WIDTH2+j] = 0;
    for (int m = 0; m < smooth_kernel_size; ++m)     // kernel rows
    {
        //int mm = MASK_WIDTH1 - 1 - m;      // row index
        int ii = i + (m - kCenterY);

        // ignore input samples which are out of bound
        if (ii >= 0 && ii < WIDTH1)
            T[i*WIDTH2+j] += N[ii*WIDTH2+j] * M[m];
    }
    //grid.sync();
    __syncthreads();

    for (int m = 0; m < smooth_kernel_size; ++m)     // kernel rows
    {
        //int mm = MASK_WIDTH1 - 1 - m;      // row index
        int jj = j + (m - kCenterX);

        // ignore input samples which are out of bound
        if (jj >= 0 && jj < WIDTH2)
            P[i*WIDTH2+j] += T[i*WIDTH2+jj] * M[m];
    }
}
int main() {

    int image[WIDTH1*WIDTH2]; // have to get some image
    int output[WIDTH1*WIDTH2]; //have to get some outtput
    double gauss_row[smooth_kernel_size];
    double sum_row = 0;
    int i, j;

    int *gpuImage, *gpuOutput;
    double *gpuT, *gpuM;

    hipMalloc(&gpuImage, WIDTH1*WIDTH2*sizeof(int));
    hipMalloc(&gpuOutput, WIDTH1*WIDTH2*sizeof(int));
    hipMalloc(&gpuT, WIDTH1*WIDTH2*sizeof(double));
    hipMalloc(&gpuM, smooth_kernel_size*sizeof(double));

    ifstream fp("/home/arun/Course_Work/EE5332/img.txt");
    if (! fp) {
        cout << "Error, file couldn't be opened" << endl; 
        return 1; 
    }    
    for(int row = 0; row < WIDTH1; row++) {  // stop loops if nothing to read
        for(int column = 0; column < WIDTH2; column++){
                fp >> image[row*WIDTH2+column];
                output[row*WIDTH2+column] = 0;
                if ( ! fp ) {
                cout << "Error reading file for element " << row << "," << column << endl; 
                return 1; 
                }
            }
    }

    for (i = 0; i < smooth_kernel_size; i++) {
        double x = i - (smooth_kernel_size - 1) / 2.0;
        gauss_row[i] = K * exp((pow(x, 2) / ((2 * pow(sigma, 2)))) * (-1));
        sum_row += gauss_row[i];
    }

    for (i = 0; i < smooth_kernel_size; i++) {
        gauss_row[i] /= sum_row;
    }

	hipMemcpy(gpuImage, image, WIDTH1*WIDTH2*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(gpuOutput, output, WIDTH1*WIDTH2*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(gpuM, gauss_row, smooth_kernel_size*sizeof(double), hipMemcpyHostToDevice);

    // for (i = 0; i < smooth_kernel_size; i++) {
    //     for (j = 0; j < smooth_kernel_size; j++) {
    //         printf("%f ", gauss[i][j]);
    //     }
    //     printf("\n");
    // }

    struct timespec begin, end; 
    clock_gettime(CLOCK_REALTIME, &begin);

    //void *args[] = {&gpuImage, &gpuM, &gpuOutput, &gpuT};
    //hipLaunchCooperativeKernel((void*)convolution_2D, dim3(WIDTH1, 1, 1), dim3(WIDTH2, 1, 1), args);
    convolution_2D<<<480, 360>>>(gpuImage, gpuM, gpuOutput, gpuT);
    hipDeviceSynchronize();

    clock_gettime(CLOCK_REALTIME, &end);
    long seconds = end.tv_sec - begin.tv_sec;
    long nanoseconds = end.tv_nsec - begin.tv_nsec;
    double elapsed = seconds + nanoseconds*1e-9;

	hipMemcpy(output, gpuOutput, WIDTH1*WIDTH2*sizeof(int), hipMemcpyDeviceToHost);
    //hipError_t err = hipGetLastError();
    //printf(“%d, %s, %s\n”, err, hipGetErrorName(err), hipGetErrorString(err))

    for(int row = 0; row < WIDTH1; row++) {  // stop loops if nothing to read
        for(int column = 0; column < WIDTH2; column++){
                cout<<output[row*WIDTH2+column]<<" ";
            }
            cout<<"\n";
    }
    printf("Time measured: %ld nanoseconds.\n", nanoseconds);
    return 0;
}
